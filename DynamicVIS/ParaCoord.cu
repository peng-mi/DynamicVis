#ifndef _PARACOORD_CU_
#define _PARACOORD_CU_

#include <GL/glew.h>
#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h> 

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/scan.h>
#include <thrust/sort.h>




#include "datamanager.h"
#include "PerformanceTimer.h"
#include "ParaCoord_kernel.cu"

 
 

//#define DEBUG_MODE
 
using namespace VIS;

#define BLOCK_SIZE 256

//variables

unsigned char   *d_buffer_ptr;
unsigned int    dn_maxDimension;
unsigned int    dn_totalBytes;
unsigned int	dn_vertexNum;
unsigned int	dn_compactItemNum;
unsigned int	dn_edgeNum;

//desc
unsigned int *d_item_offset_ptr;	
unsigned int *d_item_bytes_ptr;	
unsigned int *d_item_values_ptr;

unsigned int *d_compactValue_ptr;
unsigned int *d_selectedItems_ptr;
unsigned int dn_selectItemsNum;

//filterdata
unsigned int *d_filterData_ptr;
unsigned int  dn_filterDataNum;

//highlightData
unsigned char *d_colorValue_ptr ;
unsigned int  dn_ColorNum;
unsigned int  dn_RecordNum;
unsigned int *d_highRecord_ptr;
unsigned char *d_highIndex_ptr;



//vertex Buffer
unsigned char *d_vertex_bool_ptr;
unsigned int  *d_vertex_prefix_ptr;
unsigned int  *d_item_prefix_ptr;
unsigned int  *d_vertex_sort_ptr;

 
//edge list   
unsigned int *d_edgeDummy_ptr;
unsigned char *d_edgeBool_ptr;
unsigned int *d_edgeItemPre_ptr;
unsigned int *d_edgePrefix_ptr;

//temporal 
//unsigned char *d_boolean_vector_ptr;
//unsigned int  *d_boolean_prefix_ptr;
unsigned int  *d_dummyIndex_ptr;
unsigned int  *d_dummyValue_ptr;
unsigned char  *d_dummyBool_ptr;
unsigned int  *d_dummySum_ptr;
unsigned int  *d_compactIndex_ptr;

unsigned int  *d_item_num_ptr;
unsigned int  *d_edgeItemNum_ptr;
unsigned int  *d_vertex_Index_ptr;
unsigned char *d_colorBool_ptr;




extern "C" 
void cuda_Init()
{
	d_compactValue_ptr	=	NULL;
	d_selectedItems_ptr	=	NULL;
	
	d_filterData_ptr	=	NULL;
	d_colorValue_ptr	=	NULL;
	d_highRecord_ptr	=	NULL;
	d_highIndex_ptr		=	NULL;
	
	d_vertex_bool_ptr	=	NULL;
	d_vertex_prefix_ptr	=	NULL;
	d_item_prefix_ptr	=	NULL;
	d_vertex_sort_ptr	=	NULL;
	
	d_edgeDummy_ptr		=	NULL;
	d_edgeBool_ptr		=	NULL;
	d_edgeItemPre_ptr	=	NULL;
	d_edgePrefix_ptr	=	NULL;

	d_compactIndex_ptr	=	NULL;

	d_item_num_ptr		=	NULL;
	d_edgeItemNum_ptr	=	NULL;
	d_vertex_Index_ptr	=	NULL;
}

extern "C"
void cuda_Clean()
{
	if(d_compactValue_ptr != NULL)
		hipFree(d_compactValue_ptr);
	if(d_selectedItems_ptr != NULL)
		hipFree(d_selectedItems_ptr);
	if(d_filterData_ptr != NULL)
		hipFree(d_filterData_ptr);
	if(d_colorValue_ptr != NULL)
		hipFree(d_colorValue_ptr);
	if(d_highRecord_ptr != NULL)
		hipFree(d_highRecord_ptr);
	if(d_highIndex_ptr != NULL)
		hipFree(d_highIndex_ptr);

	if(d_vertex_bool_ptr != NULL)
		hipFree(d_vertex_bool_ptr);
	if(d_vertex_prefix_ptr != NULL)
		hipFree(d_vertex_prefix_ptr);
	if(d_item_prefix_ptr != NULL)
		hipFree(d_item_prefix_ptr);
	if(d_vertex_sort_ptr != NULL)
		hipFree(d_vertex_sort_ptr);

	if(d_edgeDummy_ptr != NULL)
		hipFree(d_edgeDummy_ptr);
	if(d_edgeBool_ptr != NULL)
		hipFree(d_edgeBool_ptr);
	if(d_edgeItemPre_ptr != NULL)
		hipFree(d_edgeItemPre_ptr);
	if(d_edgePrefix_ptr != NULL)
		hipFree(d_edgePrefix_ptr);

	if(d_compactIndex_ptr != NULL)
		hipFree(d_compactIndex_ptr);

	if(d_item_num_ptr != NULL)
		hipFree(d_item_num_ptr);
	if(d_edgeItemNum_ptr != NULL)
		hipFree(d_edgeItemNum_ptr);
	if(d_vertex_Index_ptr != NULL)
		hipFree(d_vertex_Index_ptr);

	cuda_Init();
}


extern "C"
void cuda_PrepareDataConstant(unsigned int *_item_desc_offset,	unsigned int *_item_desc_bytes,
							  unsigned int *_item_desc_values,	unsigned char *_buffer_data)
{
	d_item_offset_ptr	=	_item_desc_offset;	
	d_item_bytes_ptr	=	_item_desc_bytes;	
	d_item_values_ptr	=	_item_desc_values;
	d_buffer_ptr		=	_buffer_data;
}

extern "C"
void cuda_PrepareDataConstant2(unsigned int *_item_desc_offset,	unsigned int *_item_desc_bytes,
							   unsigned int *_item_desc_values,	unsigned char *_buffer_data,
							   unsigned int _itemNum,			unsigned int  _bufferSize)
{
#ifdef DEBUG_MODE
	RedirectIOToConsole();
#endif
	unsigned int data_size;
	unsigned int mem_size;
	data_size = _itemNum;
	mem_size = sizeof(unsigned int)*data_size;
	checkCudaErrors(hipMalloc(&d_item_offset_ptr,mem_size));
	checkCudaErrors(hipMalloc(&d_item_bytes_ptr,mem_size));
	checkCudaErrors(hipMalloc(&d_item_values_ptr,mem_size));
	checkCudaErrors(hipMemcpy( d_item_offset_ptr, _item_desc_offset, mem_size,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy( d_item_bytes_ptr, _item_desc_bytes, mem_size,hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy( d_item_values_ptr, _item_desc_values, mem_size,hipMemcpyHostToDevice));

	data_size = _bufferSize;
	mem_size = sizeof(unsigned char)*data_size;
	checkCudaErrors(hipMalloc(&d_buffer_ptr, mem_size));
	checkCudaErrors(hipMemcpy(d_buffer_ptr,_buffer_data,mem_size,hipMemcpyHostToDevice));
	hipDeviceSynchronize();

	getLastCudaError("memlloc the gpu memory 1");
}
 

extern "C"
void cuda_PrepareData(  unsigned int* _filterData,			unsigned int _filterSize,
						unsigned int *_selected_items,		unsigned int _selectedItemsNum,
						unsigned int _max_dimension,		unsigned int _total_bytes)
{


	unsigned int mem_size = 0;
	dn_maxDimension = _max_dimension;
	dn_totalBytes = _total_bytes;

	//selected items
	dn_selectItemsNum = _selectedItemsNum;
	mem_size = sizeof(unsigned int)*dn_selectItemsNum;
	if(d_selectedItems_ptr != NULL)
		hipFree(d_selectedItems_ptr);
	checkCudaErrors(hipMalloc(&d_selectedItems_ptr,mem_size));
	checkCudaErrors(hipMemcpy( d_selectedItems_ptr, _selected_items, mem_size,hipMemcpyHostToDevice));
	//filterdata
	dn_filterDataNum = _filterSize;
	mem_size = sizeof(unsigned int)*dn_filterDataNum;
	if(d_filterData_ptr != NULL)
		hipFree(d_filterData_ptr);
	checkCudaErrors(hipMalloc(&d_filterData_ptr, mem_size));
	checkCudaErrors(hipMemcpy( d_filterData_ptr, _filterData, mem_size,hipMemcpyHostToDevice));
}

extern "C"
void cuda_CleanData()
{
	if(d_selectedItems_ptr != NULL)
		hipFree(d_selectedItems_ptr);
	if(d_filterData_ptr != NULL)
		hipFree(d_filterData_ptr);
	d_selectedItems_ptr =NULL;
	d_filterData_ptr = NULL;
}

extern "C" 
void cuda_GetCompactList(unsigned int* h_compactList)
{
	//copy to the CPU
	checkCudaErrors(hipMemcpy(h_compactList,d_compactValue_ptr,sizeof(unsigned int)*dn_vertexNum,hipMemcpyDeviceToHost));
}


extern "C"
void cuda_GetCompactListSize(unsigned int &_compactNum)
{
	if(d_dummyValue_ptr != NULL)
		hipFree(d_dummyValue_ptr);
	if(d_dummyIndex_ptr != NULL)
		hipFree(d_dummyIndex_ptr);
	if(d_dummyBool_ptr != NULL)
		hipFree(d_dummyBool_ptr);
	if(d_dummySum_ptr != NULL)
		hipFree(d_dummySum_ptr);
	hipMalloc(&d_dummyValue_ptr,		sizeof(unsigned int)*dn_filterDataNum);
	hipMalloc(&d_dummyIndex_ptr,		sizeof(unsigned int)*dn_filterDataNum);
	hipMalloc(&d_dummyBool_ptr,		sizeof(unsigned char)*dn_filterDataNum);
	hipMalloc(&d_dummySum_ptr,			sizeof(unsigned int)*dn_filterDataNum);
	thrust::device_ptr<uint>			d_dummyValue(d_dummyValue_ptr);
	thrust::device_ptr<uint>			d_dummyIndex(d_dummyIndex_ptr);
	thrust::device_ptr<uint>			d_dummySum(d_dummySum_ptr);
	thrust::device_ptr<uchar>			d_dummyBool(d_dummyBool_ptr);

	 
	hipMemcpy(d_dummyIndex_ptr,	d_filterData_ptr,	sizeof(unsigned int)*dn_filterDataNum, hipMemcpyDeviceToDevice);
	//step 1 get the dummy value
	unsigned int block_num  = (dn_filterDataNum + BLOCK_SIZE -1)/BLOCK_SIZE;
	gpu_dummyValues<<<block_num, BLOCK_SIZE>>>(d_buffer_ptr,		d_item_offset_ptr,
											   d_item_bytes_ptr,	d_selectedItems_ptr,
											   d_filterData_ptr,	d_dummyValue_ptr,
											   dn_totalBytes,		dn_selectItemsNum,
											   dn_filterDataNum,	dn_maxDimension);										

 
	//step 2 sort the dummy value;
	thrust::sort_by_key(d_dummyValue, d_dummyValue+dn_filterDataNum, d_dummyIndex);
	

	 //step 3 minus the dummy value
	block_num = (dn_filterDataNum +  BLOCK_SIZE -1)/BLOCK_SIZE;
	gpu_minusDummyValue<<< block_num, BLOCK_SIZE >>>(d_dummyValue_ptr, d_dummyBool_ptr, dn_filterDataNum);
	
	//step 4 scan the dummy value
	thrust::exclusive_scan(d_dummyBool, d_dummyBool+dn_filterDataNum,d_dummySum);
	
	//step 5 compact the dummy value
	dn_compactItemNum = d_dummySum[dn_filterDataNum-1];
	if (d_dummyBool[dn_filterDataNum-1] == 1)
		dn_compactItemNum = dn_compactItemNum+1;

#ifdef DEBUG_MODE
	fprintf(stdout, "compactItemNum, %d\n",dn_compactItemNum);
	fprintf(stdout, "hello world\n");
#endif

	_compactNum = dn_compactItemNum;
	if(d_compactIndex_ptr != NULL)
		hipFree(d_compactIndex_ptr);
	if(d_compactValue_ptr)
		hipFree(d_compactValue_ptr);
	hipMalloc(&d_compactIndex_ptr,		sizeof(unsigned int)*dn_compactItemNum);
	hipMalloc(&d_compactValue_ptr,		sizeof(unsigned int)*dn_compactItemNum*dn_selectItemsNum);
	thrust::device_ptr<unsigned int>	d_compactIndex(d_compactIndex_ptr);
	thrust::device_ptr<unsigned int>	d_compactValue(d_compactValue_ptr);

	block_num  = (dn_filterDataNum + BLOCK_SIZE -1)/BLOCK_SIZE;
	gpu_compactValue<<< block_num, BLOCK_SIZE >>>(d_buffer_ptr,			d_item_offset_ptr,
												  d_item_bytes_ptr,		d_dummyIndex_ptr,
												  d_selectedItems_ptr,	d_dummyBool_ptr,
												  d_dummySum_ptr,		d_compactValue_ptr,
												  d_compactIndex_ptr,	dn_totalBytes,
												  dn_filterDataNum,		dn_selectItemsNum,
												  dn_compactItemNum);
	

#ifdef DEBUG_MODE
	thrust::host_vector<uint> H3(dn_compactItemNum*dn_selectItemsNum,0);
	thrust::host_vector<uint> H4(dn_compactItemNum,0);
	thrust::copy(d_compactValue,d_compactValue+dn_compactItemNum*dn_selectItemsNum,H3.begin());
	thrust::copy(d_compactIndex,d_compactIndex+dn_compactItemNum,H4.begin());
	fprintf(stdout, "\n Compact values:\n");
	for(uint i=0; i<dn_compactItemNum*dn_selectItemsNum; i++)
		fprintf(stdout, "%d	",H3[i]);

	fprintf(stdout, "\n Compact Index:\n");
	for(uint i=0; i<dn_compactItemNum; i++)
		fprintf(stdout, "%d	",H4[i]);

	fprintf(stdout, "hello world");
#endif
}

extern "C"
void cuda_GetVertexIndexSize(unsigned int &vertexBufferSize,	unsigned int &edgeBufferSize)
{
	//step 6 vertex buffer
	if(d_vertex_bool_ptr != NULL)
		hipFree(d_vertex_bool_ptr);
	if(d_vertex_prefix_ptr != NULL)
		hipFree(d_vertex_prefix_ptr);
	if(d_vertex_sort_ptr != NULL)
		hipFree(d_vertex_sort_ptr);
	if(d_item_prefix_ptr != NULL)
		hipFree(d_item_prefix_ptr);
	if(d_item_num_ptr != NULL)
		hipFree(d_item_num_ptr);
	dn_vertexNum  = dn_selectItemsNum*dn_compactItemNum;
	hipMalloc(&d_vertex_bool_ptr,		sizeof(unsigned char)*dn_vertexNum);
	hipMalloc(&d_vertex_prefix_ptr,	sizeof(unsigned int)*dn_vertexNum);
	hipMalloc(&d_vertex_sort_ptr,		sizeof(unsigned int)*dn_vertexNum);
	hipMalloc(&d_item_prefix_ptr,		sizeof(unsigned int)*dn_selectItemsNum);
	hipMalloc(&d_item_num_ptr,			sizeof(unsigned int)*dn_selectItemsNum);
	thrust::device_ptr<unsigned char>	d_vertex_bool(d_vertex_bool_ptr);
	thrust::device_ptr<unsigned int>	d_vertex_prefix(d_vertex_prefix_ptr);
	thrust::device_ptr<unsigned int>	d_vertex_sort(d_vertex_sort_ptr);
	thrust::device_ptr<unsigned int>	d_item_prefix(d_item_prefix_ptr);
	thrust::device_ptr<unsigned int>	d_item_num(d_item_num_ptr);
	hipMemcpy(d_vertex_sort_ptr, d_compactValue_ptr, sizeof(unsigned int)*dn_vertexNum, hipMemcpyDeviceToDevice);
	
	//step 7 sort each selected items;
	for(unsigned int i =0; i< dn_selectItemsNum ; i++)
		thrust::sort(d_vertex_sort+i*dn_compactItemNum,d_vertex_sort+(i+1)*dn_compactItemNum);
	
	
	//step 8 get the bool of vertex
	block_num  = (dn_vertexNum + BLOCK_SIZE -1)/BLOCK_SIZE;
	gpu_vertexBool<<< block_num, BLOCK_SIZE >>>(d_vertex_sort_ptr,		d_vertex_bool_ptr,
												dn_compactItemNum,		dn_vertexNum);

#ifdef DEBUG_MODE
	thrust::host_vector<uchar> H5(dn_vertexNum,0);
	thrust::copy(d_vertex_bool,d_vertex_bool+dn_vertexNum,H5.begin());
	fprintf(stdout, "\nhello world\n");
	for(uint i=0; i<dn_vertexNum; i++)
		fprintf(stdout,"%d	",H5[i]);
	fprintf(stdout, "vertex buffer size %d\n",vertexBufferSize);
	fprintf(stdout, "\nhello world\n");
#endif

	//step 9 exclusive scan of the vertex
	vertexBufferSize  = 0;
	d_item_prefix[0] = 0;
	for( unsigned int i =0; i<dn_selectItemsNum; i++)
	{
		thrust::exclusive_scan(d_vertex_bool+i*dn_compactItemNum,
							   d_vertex_bool+(i+1)*dn_compactItemNum,
							   d_vertex_prefix+i*dn_compactItemNum);
		d_item_num[i] = d_vertex_prefix[(i+1)*dn_compactItemNum-1];
		if(d_vertex_bool[(i+1)*dn_compactItemNum-1] == 1)
			d_item_num[i] = d_item_num[i] +1;
		vertexBufferSize += d_item_num[i];

		if (i!=0)
			d_item_prefix[i] = d_item_prefix[i-1] +  d_item_num[i-1]; 
	}

#ifdef DEBUG_MODE
	fprintf(stdout, "vertex buffer size %d\n",vertexBufferSize);
	fprintf(stdout, "hello world");
#endif
	

	//step 10 the edge info
	dn_edgeNum  = dn_compactItemNum*(dn_selectItemsNum-1);
	if(d_edgeBool_ptr != NULL)
		hipFree(d_edgeBool_ptr);
	if(d_edgeDummy_ptr != NULL)
		hipFree(d_edgeDummy_ptr);
	if(d_edgePrefix_ptr != NULL)
		hipFree(d_edgePrefix_ptr);
	if(d_edgeItemPre_ptr != NULL)
		hipFree(d_edgeItemPre_ptr);
	if(d_edgeItemNum_ptr != NULL)
		hipFree(d_edgeItemNum_ptr);

	hipMalloc(&d_edgeBool_ptr,			sizeof(unsigned char)*dn_edgeNum);
	hipMalloc(&d_edgeDummy_ptr,		sizeof(unsigned int)*dn_edgeNum);
	hipMalloc(&d_edgePrefix_ptr,		sizeof(unsigned int)*dn_edgeNum);
	hipMalloc(&d_edgeItemPre_ptr,		sizeof(unsigned int)*(dn_selectItemsNum-1));
	hipMalloc(&d_edgeItemNum_ptr,		sizeof(unsigned int)*(dn_selectItemsNum-1));
	thrust::device_ptr<unsigned char>	d_edgeBool(d_edgeBool_ptr);
	thrust::device_ptr<unsigned int>	d_edgeDummy(d_edgeDummy_ptr);
	thrust::device_ptr<unsigned int>	d_edgePrefix(d_edgePrefix_ptr);
	thrust::device_ptr<unsigned int>	d_edgeItemPre(d_edgeItemPre_ptr);
	thrust::device_ptr<unsigned int>	d_edgeItemNum(d_edgeItemNum_ptr);
 
	block_num  = (dn_edgeNum + BLOCK_SIZE-1)/BLOCK_SIZE;
	gpu_edgeDummyValue<<< block_num, BLOCK_SIZE >>>(d_compactValue_ptr,		d_edgeDummy_ptr,
													dn_edgeNum,				dn_compactItemNum,
													dn_maxDimension,		dn_selectItemsNum);

	//step 11
	for ( unsigned int i = 0; i< dn_selectItemsNum-1; i++)
	{
		thrust::sort(d_edgeDummy + i*dn_compactItemNum,
					d_edgeDummy + (i+1)*dn_compactItemNum);
	}
	
	//step 12
	gpu_edgeBool<<< block_num, BLOCK_SIZE >>>(d_edgeDummy_ptr,		d_edgeBool_ptr,
											  dn_edgeNum,			dn_compactItemNum);


	edgeBufferSize = 0;
	d_edgeItemPre[0] = 0;
	for (unsigned int i =0; i < dn_selectItemsNum -1; i++)
	{
		thrust::exclusive_scan(d_edgeBool +  i*dn_compactItemNum,
								d_edgeBool + (i+1)*dn_compactItemNum,
								d_edgePrefix +i*dn_compactItemNum);

		d_edgeItemNum[i] = d_edgePrefix[(i+1)*dn_compactItemNum-1];
		if (d_edgeBool[(i+1)*dn_compactItemNum-1] == 1)
			d_edgeItemNum[i]  = d_edgeItemNum[i] +1;
		edgeBufferSize += d_edgeItemNum[i];

		if( i !=0)
			d_edgeItemPre[i] = d_edgeItemPre[i-1] + d_edgeItemNum[i-1];
	}

	
#ifdef DEBUG_MODE
	fprintf(stdout, "\n vertex buffer size %d\n",edgeBufferSize);
	fprintf(stdout, "hello world");
#endif

	

	//step 13 clean the unrelated vectors
	hipFree(d_dummyValue_ptr);
	hipFree(d_dummyIndex_ptr);
	hipFree(d_dummyBool_ptr);
	hipFree(d_dummySum_ptr);
	d_dummyValue_ptr = NULL;
	d_dummyIndex_ptr = NULL;
	d_dummyBool_ptr = NULL;
	d_dummySum_ptr = NULL;
}



extern "C"
void cuda_GenerateSize(unsigned int &vertexBufferSize,	unsigned int &edgeBufferSize,
					   unsigned int &_compactNum)
{
	if(d_dummyValue_ptr != NULL)
		hipFree(d_dummyValue_ptr);
	if(d_dummyIndex_ptr != NULL)
		hipFree(d_dummyIndex_ptr);
	if(d_dummyBool_ptr != NULL)
		hipFree(d_dummyBool_ptr);
	if(d_dummySum_ptr != NULL)
		hipFree(d_dummySum_ptr);
	hipMalloc(&d_dummyValue_ptr,		sizeof(unsigned int)*dn_filterDataNum);
	hipMalloc(&d_dummyIndex_ptr,		sizeof(unsigned int)*dn_filterDataNum);
	hipMalloc(&d_dummyBool_ptr,		sizeof(unsigned char)*dn_filterDataNum);
	hipMalloc(&d_dummySum_ptr,			sizeof(unsigned int)*dn_filterDataNum);
	thrust::device_ptr<uint>			d_dummyValue(d_dummyValue_ptr);
	thrust::device_ptr<uint>			d_dummyIndex(d_dummyIndex_ptr);
	thrust::device_ptr<uint>			d_dummySum(d_dummySum_ptr);
	thrust::device_ptr<uchar>			d_dummyBool(d_dummyBool_ptr);

	 
	hipMemcpy(d_dummyIndex_ptr,	d_filterData_ptr,	sizeof(unsigned int)*dn_filterDataNum, hipMemcpyDeviceToDevice);
	//step 1 get the dummy value
	unsigned int block_num  = (dn_filterDataNum + BLOCK_SIZE -1)/BLOCK_SIZE;
	gpu_dummyValues<<<block_num, BLOCK_SIZE>>>(d_buffer_ptr,		d_item_offset_ptr,
											   d_item_bytes_ptr,	d_selectedItems_ptr,
											   d_filterData_ptr,	d_dummyValue_ptr,
											   dn_totalBytes,		dn_selectItemsNum,
											   dn_filterDataNum,	dn_maxDimension);										
	

#ifdef DEBUG_MODE
	fprintf(stdout, "num of data %d , max_dimension, %d\n",dn_filterDataNum,dn_maxDimension);
	thrust::host_vector<uint> H1(dn_filterDataNum,0);
	thrust::host_vector<uint> H2(dn_filterDataNum,0);
	thrust::copy(d_dummyValue,d_dummyValue+dn_filterDataNum,H1.begin());
	thrust::copy(d_dummyIndex,d_dummyIndex+dn_filterDataNum,H2.begin());
	for(uint i=0; i< dn_filterDataNum; i++)
		fprintf(stdout,"%d \n",H1[i]);

	for(uint i=0; i< dn_filterDataNum; i++)
		fprintf(stdout,"%d \n",H2[i]);
#endif

	//step 2 sort the dummy value;
	thrust::sort_by_key(d_dummyValue, d_dummyValue+dn_filterDataNum, d_dummyIndex);

	

	 //step 3 minus the dummy value
	block_num = (dn_filterDataNum +  BLOCK_SIZE -1)/BLOCK_SIZE;
	gpu_minusDummyValue<<< block_num, BLOCK_SIZE >>>(d_dummyValue_ptr, d_dummyBool_ptr, dn_filterDataNum);
	hipDeviceSynchronize();

	//step 4 scan the dummy value
	thrust::exclusive_scan(d_dummyBool, d_dummyBool+dn_filterDataNum,d_dummySum);
	hipDeviceSynchronize();

	//step 5 compact the dummy value
	dn_compactItemNum = d_dummySum[dn_filterDataNum-1];
	if (d_dummyBool[dn_filterDataNum-1] == 1)
		dn_compactItemNum = dn_compactItemNum+1;

#ifdef DEBUG_MODE
	fprintf(stdout, "compactItemNum, %d\n",dn_compactItemNum);
	fprintf(stdout, "hello world\n");
#endif

	_compactNum = dn_compactItemNum;
	if(d_compactIndex_ptr != NULL)
		hipFree(d_compactIndex_ptr);
	if(d_compactValue_ptr)
		hipFree(d_compactValue_ptr);
	hipMalloc(&d_compactIndex_ptr,		sizeof(unsigned int)*dn_compactItemNum);
	hipMalloc(&d_compactValue_ptr,		sizeof(unsigned int)*dn_compactItemNum*dn_selectItemsNum);
	thrust::device_ptr<unsigned int>	d_compactIndex(d_compactIndex_ptr);
	thrust::device_ptr<unsigned int>	d_compactValue(d_compactValue_ptr);

	block_num  = (dn_filterDataNum + BLOCK_SIZE -1)/BLOCK_SIZE;
	gpu_compactValue<<< block_num, BLOCK_SIZE >>>(d_buffer_ptr,			d_item_offset_ptr,
												  d_item_bytes_ptr,		d_dummyIndex_ptr,
												  d_selectedItems_ptr,	d_dummyBool_ptr,
												  d_dummySum_ptr,		d_compactValue_ptr,
												  d_compactIndex_ptr,	dn_totalBytes,
												  dn_filterDataNum,		dn_selectItemsNum,
												  dn_compactItemNum);
	hipDeviceSynchronize();


#ifdef DEBUG_MODE
	thrust::host_vector<uint> H3(dn_compactItemNum*dn_selectItemsNum,0);
	thrust::host_vector<uint> H4(dn_compactItemNum,0);
	thrust::copy(d_compactValue,d_compactValue+dn_compactItemNum*dn_selectItemsNum,H3.begin());
	thrust::copy(d_compactIndex,d_compactIndex+dn_compactItemNum,H4.begin());
	fprintf(stdout, "\n Compact values:\n");
	for(uint i=0; i<dn_compactItemNum*dn_selectItemsNum; i++)
		fprintf(stdout, "%d	",H3[i]);

	fprintf(stdout, "\n Compact Index:\n");
	for(uint i=0; i<dn_compactItemNum; i++)
		fprintf(stdout, "%d	",H4[i]);

	fprintf(stdout, "hello world");
#endif


	//step 6 vertex buffer
	if(d_vertex_bool_ptr != NULL)
		hipFree(d_vertex_bool_ptr);
	if(d_vertex_prefix_ptr != NULL)
		hipFree(d_vertex_prefix_ptr);
	if(d_vertex_sort_ptr != NULL)
		hipFree(d_vertex_sort_ptr);
	if(d_item_prefix_ptr != NULL)
		hipFree(d_item_prefix_ptr);
	if(d_item_num_ptr != NULL)
		hipFree(d_item_num_ptr);
	dn_vertexNum  = dn_selectItemsNum*dn_compactItemNum;
	hipMalloc(&d_vertex_bool_ptr,		sizeof(unsigned char)*dn_vertexNum);
	hipMalloc(&d_vertex_prefix_ptr,	sizeof(unsigned int)*dn_vertexNum);
	hipMalloc(&d_vertex_sort_ptr,		sizeof(unsigned int)*dn_vertexNum);
	hipMalloc(&d_item_prefix_ptr,		sizeof(unsigned int)*dn_selectItemsNum);
	hipMalloc(&d_item_num_ptr,			sizeof(unsigned int)*dn_selectItemsNum);
	thrust::device_ptr<unsigned char>	d_vertex_bool(d_vertex_bool_ptr);
	thrust::device_ptr<unsigned int>	d_vertex_prefix(d_vertex_prefix_ptr);
	thrust::device_ptr<unsigned int>	d_vertex_sort(d_vertex_sort_ptr);
	thrust::device_ptr<unsigned int>	d_item_prefix(d_item_prefix_ptr);
	thrust::device_ptr<unsigned int>	d_item_num(d_item_num_ptr);
	hipMemcpy(d_vertex_sort_ptr, d_compactValue_ptr, sizeof(unsigned int)*dn_vertexNum, hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();

	//step 7 sort each selected items;
	for(unsigned int i =0; i< dn_selectItemsNum ; i++)
	{
		thrust::sort(d_vertex_sort+i*dn_compactItemNum,d_vertex_sort+(i+1)*dn_compactItemNum);
		hipDeviceSynchronize();
	}
	
	//step 8 get the bool of vertex
	block_num  = (dn_vertexNum + BLOCK_SIZE -1)/BLOCK_SIZE;
	gpu_vertexBool<<< block_num, BLOCK_SIZE >>>(d_vertex_sort_ptr,		d_vertex_bool_ptr,
												dn_compactItemNum,		dn_vertexNum);
	hipDeviceSynchronize();

#ifdef DEBUG_MODE
	thrust::host_vector<uchar> H5(dn_vertexNum,0);
	thrust::copy(d_vertex_bool,d_vertex_bool+dn_vertexNum,H5.begin());
	fprintf(stdout, "\nhello world\n");
	for(uint i=0; i<dn_vertexNum; i++)
		fprintf(stdout,"%d	",H5[i]);
	fprintf(stdout, "vertex buffer size %d\n",vertexBufferSize);
	fprintf(stdout, "\nhello world\n");
#endif

	//step 9 exclusive scan of the vertex
	vertexBufferSize  = 0;
	d_item_prefix[0] = 0;
	for( unsigned int i =0; i<dn_selectItemsNum; i++)
	{
		thrust::exclusive_scan(d_vertex_bool+i*dn_compactItemNum,
							   d_vertex_bool+(i+1)*dn_compactItemNum,
							   d_vertex_prefix+i*dn_compactItemNum);
		d_item_num[i] = d_vertex_prefix[(i+1)*dn_compactItemNum-1];
		if(d_vertex_bool[(i+1)*dn_compactItemNum-1] == 1)
			d_item_num[i] = d_item_num[i] +1;
		vertexBufferSize += d_item_num[i];

		if (i!=0)
			d_item_prefix[i] = d_item_prefix[i-1] +  d_item_num[i-1]; 
		hipDeviceSynchronize();
	}

#ifdef DEBUG_MODE
	fprintf(stdout, "vertex buffer size %d\n",vertexBufferSize);
	fprintf(stdout, "hello world");
#endif
	

	//step 10 the edge info
	dn_edgeNum  = dn_compactItemNum*(dn_selectItemsNum-1);
	if(d_edgeBool_ptr != NULL)
		hipFree(d_edgeBool_ptr);
	if(d_edgeDummy_ptr != NULL)
		hipFree(d_edgeDummy_ptr);
	if(d_edgePrefix_ptr != NULL)
		hipFree(d_edgePrefix_ptr);
	if(d_edgeItemPre_ptr != NULL)
		hipFree(d_edgeItemPre_ptr);
	if(d_edgeItemNum_ptr != NULL)
		hipFree(d_edgeItemNum_ptr);

	hipMalloc(&d_edgeBool_ptr,			sizeof(unsigned char)*dn_edgeNum);
	hipMalloc(&d_edgeDummy_ptr,		sizeof(unsigned int)*dn_edgeNum);
	hipMalloc(&d_edgePrefix_ptr,		sizeof(unsigned int)*dn_edgeNum);
	hipMalloc(&d_edgeItemPre_ptr,		sizeof(unsigned int)*(dn_selectItemsNum-1));
	hipMalloc(&d_edgeItemNum_ptr,		sizeof(unsigned int)*(dn_selectItemsNum-1));
	thrust::device_ptr<unsigned char>	d_edgeBool(d_edgeBool_ptr);
	thrust::device_ptr<unsigned int>	d_edgeDummy(d_edgeDummy_ptr);
	thrust::device_ptr<unsigned int>	d_edgePrefix(d_edgePrefix_ptr);
	thrust::device_ptr<unsigned int>	d_edgeItemPre(d_edgeItemPre_ptr);
	thrust::device_ptr<unsigned int>	d_edgeItemNum(d_edgeItemNum_ptr);
 
	block_num  = (dn_edgeNum + BLOCK_SIZE-1)/BLOCK_SIZE;
	gpu_edgeDummyValue<<< block_num, BLOCK_SIZE >>>(d_compactValue_ptr,		d_edgeDummy_ptr,
													dn_edgeNum,				dn_compactItemNum,
													dn_maxDimension,		dn_selectItemsNum);
	hipDeviceSynchronize();

	//step 11
	for ( unsigned int i = 0; i< dn_selectItemsNum-1; i++)
	{
		thrust::sort(d_edgeDummy + i*dn_compactItemNum,
					d_edgeDummy + (i+1)*dn_compactItemNum);
		hipDeviceSynchronize();
	}
	
	//step 12
	gpu_edgeBool<<< block_num, BLOCK_SIZE >>>(d_edgeDummy_ptr,		d_edgeBool_ptr,
											  dn_edgeNum,			dn_compactItemNum);
	hipDeviceSynchronize();


	edgeBufferSize = 0;
	d_edgeItemPre[0] = 0;
	for (unsigned int i =0; i < dn_selectItemsNum -1; i++)
	{
		thrust::exclusive_scan(d_edgeBool +  i*dn_compactItemNum,
								d_edgeBool + (i+1)*dn_compactItemNum,
								d_edgePrefix +i*dn_compactItemNum);

		d_edgeItemNum[i] = d_edgePrefix[(i+1)*dn_compactItemNum-1];
		if (d_edgeBool[(i+1)*dn_compactItemNum-1] == 1)
			d_edgeItemNum[i]  = d_edgeItemNum[i] +1;
		edgeBufferSize += d_edgeItemNum[i];

		if( i !=0)
			d_edgeItemPre[i] = d_edgeItemPre[i-1] + d_edgeItemNum[i-1];
		hipDeviceSynchronize();
	}

	
#ifdef DEBUG_MODE
	fprintf(stdout, "\n vertex buffer size %d\n",edgeBufferSize);
	fprintf(stdout, "hello world");
#endif

	

	//step 13 clean the unrelated vectors
	hipFree(d_dummyValue_ptr);
	hipFree(d_dummyIndex_ptr);
	hipFree(d_dummyBool_ptr);
	hipFree(d_dummySum_ptr);
	d_dummyValue_ptr = NULL;
	d_dummyIndex_ptr = NULL;
	d_dummyBool_ptr = NULL;
	d_dummySum_ptr = NULL;

}

 

extern "C"
void cuda_GenerateVertexBuffers(float* d_vertexBuffer_ptr,			unsigned int *d_edgeBuffer_ptr,
								//unsigned char* d_colorBuffer_ptr,	
								unsigned int vertexBufferSize, 
								unsigned int edgeBufferSize)
{

	//set the default color buffer
	//checkCudaErrors(hipMemset(d_colorBuffer_ptr,128,sizeof(unsigned char)*vertexBufferSize*4));
	

	if(d_vertex_Index_ptr != NULL)
		hipFree(d_vertex_Index_ptr);
	hipMalloc(&d_vertex_Index_ptr,		sizeof(unsigned int)*dn_vertexNum);
	thrust::device_ptr<unsigned int>	d_vertex_Index(d_vertex_Index_ptr);

	unsigned int block_num  = (dn_vertexNum + BLOCK_SIZE -1)/BLOCK_SIZE;
	gpu_generateVertexBuffer<<< block_num, BLOCK_SIZE >>>(d_vertex_bool_ptr,		d_vertex_prefix_ptr,
														  d_vertex_Index_ptr,		d_vertex_sort_ptr,
														  d_item_prefix_ptr,		d_item_values_ptr,
														  d_selectedItems_ptr,		d_vertexBuffer_ptr,
														  dn_vertexNum,				dn_compactItemNum);
	hipDeviceSynchronize();

#ifdef DEBUG_MODE
	fprintf(stdout, "\n \n Vertex Index \n\n");
	thrust::host_vector<uchar> H1(dn_vertexNum,0);
	thrust::copy(d_vertex_Index,d_vertex_Index+dn_vertexNum,H1.begin());
	fprintf(stdout, "\nhello world\n");
	for(uint i=0; i<dn_vertexNum; i++)
		fprintf(stdout,"%d	",H1[i]);
	fprintf(stdout, "\n vertex index %d\n",dn_vertexNum);
	fprintf(stdout, "hello world");
#endif

	//generate EdgeBuffer
	block_num  = (dn_edgeNum + BLOCK_SIZE-1)/BLOCK_SIZE;
	gpu_generateEdgeBuffer<<< block_num, BLOCK_SIZE >>>(d_vertex_Index_ptr,		d_edgeBuffer_ptr,
														d_edgeDummy_ptr,		d_edgeBool_ptr,
														d_edgeItemPre_ptr,		d_edgePrefix_ptr,
														d_item_prefix_ptr,		dn_vertexNum,
														vertexBufferSize,		dn_compactItemNum,
														dn_edgeNum,				dn_selectItemsNum,
														dn_maxDimension);
	hipDeviceSynchronize();

#ifdef DEBUG_MODE
	fprintf(stdout, "\nhello world\n");
	thrust::device_ptr<unsigned int> d_edgeBuffer(d_edgeBuffer_ptr);
	thrust::host_vector<uchar> H2(dn_vertexNum*2,0);
	thrust::copy(d_edgeBuffer,d_edgeBuffer+dn_edgeNum*2,H2.begin());
	fprintf(stdout, "\nhello world\n");
	for(uint i=0; i<dn_edgeNum*2; i++)
		fprintf(stdout,"%d	",H2[i]);
	fprintf(stdout, "\n vertex index %d\n",dn_edgeNum);
	fprintf(stdout, "hello world");


	fprintf(stdout, "\n \n Vertex Index \n\n");
	thrust::copy(d_vertex_Index,d_vertex_Index+dn_vertexNum,H1.begin());
	fprintf(stdout, "\nhello world\n");
	for(uint i=0; i<dn_vertexNum; i++)
		fprintf(stdout,"%d	",H1[i]);
	fprintf(stdout, "\n vertex index %d\n",dn_vertexNum);
	fprintf(stdout, "hello world");

#endif
}

extern "C"
void cuda_PrepareColorData(unsigned char* _colorValue,	unsigned int* _highRecord,
							   unsigned char* _highIndex,	
							   unsigned int _numColor,		unsigned int  _numRecord)
{
	dn_ColorNum =  _numColor;
	dn_RecordNum = _numRecord;

	unsigned int mem_size = 0;
	mem_size = sizeof(unsigned char)*dn_ColorNum*4;
	if(d_colorValue_ptr != NULL)
		hipFree(d_colorValue_ptr);
	checkCudaErrors(hipMalloc(&d_colorValue_ptr,mem_size));
	checkCudaErrors(hipMemcpy( d_colorValue_ptr, _colorValue, mem_size,hipMemcpyHostToDevice));
	hipDeviceSynchronize();

	mem_size = sizeof(unsigned int)*dn_RecordNum;
	if(d_highRecord_ptr != NULL)
		hipFree(d_highRecord_ptr);
	hipMalloc(&d_highRecord_ptr,mem_size);
	hipMemcpy( d_highRecord_ptr, _highRecord, mem_size,hipMemcpyHostToDevice);
	cutilDeviceSynchronize();

	mem_size = sizeof(unsigned char)*dn_RecordNum;
	if(d_highIndex_ptr != NULL)
		hipFree(d_highIndex_ptr);
	hipMalloc(&d_highIndex_ptr,mem_size);
	hipMemcpy( d_highIndex_ptr, _highIndex, mem_size,hipMemcpyHostToDevice);
	cutilDeviceSynchronize();

#ifdef DEBUG_MODE
	fprintf(stdout, "\nhello world\n");
	fprintf(stdout,"number of color %d, number of records: %d\n",dn_ColorNum,dn_RecordNum);
	thrust::host_vector<uchar> H1(dn_RecordNum,0);
	thrust::device_ptr<unsigned int> d_highRecord(d_highRecord_ptr);
	thrust::copy(d_highRecord,d_highRecord+dn_RecordNum,H1.begin());
	for(uint i=0; i<dn_RecordNum; i++)
		fprintf(stdout,"%d	",H1[i]);
#endif

}

extern "C"
void cuda_CleanColorData()
{
	if(d_colorValue_ptr != NULL)
		hipFree(d_colorValue_ptr);
	if(d_highRecord_ptr != NULL)
		hipFree(d_highRecord_ptr);
	if(d_highIndex_ptr != NULL)
		hipFree(d_highIndex_ptr);
	d_colorValue_ptr = NULL;
	d_highRecord_ptr = NULL;
	d_highIndex_ptr = NULL;
}

extern "C"
void cuda_SetDefaultColorBuffer(unsigned char* d_colorBuffer_ptr, unsigned int _colorBufferSize)
{
	unsigned int mem_size = _colorBufferSize*4*sizeof(unsigned char);
	checkCudaErrors((hipMemset(d_colorBuffer_ptr,179,mem_size)));
}


extern "C"
void cuda_GenerateColorBuffers(unsigned char* d_colorBuffer_ptr, unsigned int _colorBufferSize)
{
	uint block_num = (dn_RecordNum + BLOCK_SIZE-1)/BLOCK_SIZE;

	if(d_dummyValue_ptr != NULL)
		hipFree(d_dummyValue_ptr);
	checkCudaErrors(hipMalloc(&d_dummyValue_ptr,		sizeof(unsigned int)*dn_RecordNum));
	hipMemset(d_dummyValue_ptr,0,sizeof(unsigned int)*dn_RecordNum);

	//step 1 get the dummy value
	gpu_dummyValues<<<block_num, BLOCK_SIZE>>>( d_buffer_ptr,			d_item_offset_ptr,
											   	d_item_bytes_ptr,		d_selectedItems_ptr,
												d_highRecord_ptr,		d_dummyValue_ptr,
												dn_totalBytes,			dn_selectItemsNum,
												dn_RecordNum,			dn_maxDimension);										
	hipDeviceSynchronize();

#ifdef DEBUG_MODE
	fprintf(stdout, "\Dummy value\n");
	fprintf(stdout,"number of color %d, number of records: %d, maxdimension:%d selectedItems %d, totalBytes:%d\n",
		dn_ColorNum,dn_RecordNum,dn_maxDimension,dn_selectItemsNum,dn_totalBytes);
	thrust::host_vector<uint> H1(dn_RecordNum,0);
	thrust::device_ptr<unsigned int> d_dummyValue11(d_dummyValue_ptr);
	thrust::copy(d_dummyValue11,d_dummyValue11+dn_RecordNum,H1.begin());
	for(uint i=0; i<dn_RecordNum; i++)
		fprintf(stdout,"%d	",H1[i]);
#endif

	//step 2 sort the dummy value
	thrust::device_ptr<unsigned int> d_dummyValue(d_dummyValue_ptr);
	thrust::device_ptr<unsigned char> d_highIndex(d_highIndex_ptr);
	thrust::sort_by_key(d_dummyValue,d_dummyValue+dn_RecordNum,d_highIndex);
	cutilDeviceSynchronize();

#ifdef DEBUG_MODE
	fprintf(stdout, "\Sort Dummy value\n");
	fprintf(stdout,"number of color %d, number of records: %d\n",dn_ColorNum,dn_RecordNum);
	thrust::host_vector<uint> H2(dn_RecordNum,0);
	thrust::device_ptr<unsigned int> d_dummyValue22(d_dummyValue_ptr);
	thrust::copy(d_dummyValue22,d_dummyValue22+dn_RecordNum,H2.begin());
	for(uint i=0; i<dn_RecordNum; i++)
		fprintf(stdout,"%d	",H2[i]);
#endif

	//step 3 mark the compact item list
	 
	block_num = (dn_compactItemNum + BLOCK_SIZE-1)/BLOCK_SIZE;
	if(d_colorBool_ptr != NULL)
		hipFree(d_colorBool_ptr);
	checkCudaErrors(hipMalloc(&d_colorBool_ptr,		sizeof(unsigned char)*dn_compactItemNum*dn_selectItemsNum));
	
	gpuColorFilter<<<block_num, BLOCK_SIZE>>>(d_compactValue_ptr,		d_dummyValue_ptr,
											  d_highIndex_ptr,			d_colorBool_ptr,
											  dn_compactItemNum,		dn_RecordNum,
											  dn_selectItemsNum,		dn_maxDimension);
	hipDeviceSynchronize();

#ifdef DEBUG_MODE
	fprintf(stdout, "\n Color Filter\n");
	fprintf(stdout,"number of compact values %d\n",dn_compactItemNum);
 	thrust::host_vector<uchar> H3(dn_compactItemNum,0);

 	 

	thrust::host_vector<uint> H7(dn_compactItemNum*dn_selectItemsNum,0);
	thrust::device_ptr<unsigned int> d_compactValuesd(d_compactValue_ptr);
	thrust::copy(d_compactValuesd,d_compactValuesd+dn_compactItemNum*dn_selectItemsNum,H7.begin());
	for(uint j=0; j<dn_selectItemsNum; j++)
	{
		fprintf(stdout,"\n");
		for(uint i=0; i<dn_compactItemNum; i++)
			fprintf(stdout,"%d	",H7[i+j*dn_selectItemsNum]);
	}

 

	thrust::device_ptr<unsigned char> d_colorBool(d_colorBool_ptr);
	thrust::copy(d_colorBool,d_colorBool+dn_compactItemNum,H3.begin());
	for(uint i=0; i<dn_compactItemNum; i++)
		fprintf(stdout,"%d	",H3[i]);
#endif


 
	unsigned int mem_size = _colorBufferSize*4*sizeof(unsigned char);
	checkCudaErrors((hipMemset(d_colorBuffer_ptr,179,mem_size)));
	 
	//step 4 fill the color Buffer
	block_num = (dn_compactItemNum + BLOCK_SIZE-1)/BLOCK_SIZE;
	gpuColorBuffer<<<block_num, BLOCK_SIZE>>>(d_vertex_Index_ptr,		d_highIndex_ptr,
											  d_item_prefix_ptr,		d_compactValue_ptr,
											  d_colorValue_ptr,			d_colorBool_ptr,
											  d_colorBuffer_ptr,		dn_vertexNum,
											  dn_compactItemNum,		dn_selectItemsNum,
											  _colorBufferSize);
	hipDeviceSynchronize();
	//checkCudaErrors((hipMemset(d_colorBuffer_ptr,128,mem_size)));

#ifdef DEBUG_MODE
	fprintf(stdout, "\Color buffer\n");
	fprintf(stdout,"color buffer size %d\n",_colorBufferSize);



	fprintf(stdout,"\n vertex size %d\n",dn_vertexNum);
	thrust::host_vector<uint> H12(dn_vertexNum,0);
	thrust::device_ptr<unsigned int> d_vert(d_vertex_Index_ptr);
	thrust::copy(d_vert,d_vert+dn_vertexNum,H12.begin());
	for(uint i=0; i<dn_vertexNum; i++)
		fprintf(stdout,"%d	",H12[i]);

	 

#endif 
	 

	//step 5 free space
	hipFree(d_dummyValue_ptr);
	//hipFree(d_dummyIndex_ptr);
	hipFree(d_colorBool_ptr);
	d_dummyValue_ptr = NULL;
	d_colorBool_ptr = NULL;

}

#endif