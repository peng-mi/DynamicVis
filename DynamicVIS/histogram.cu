//thrust
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/functional.h>

extern "C"
void cuda_HistogramOrder(unsigned int* _histogramData, unsigned int* _reference, unsigned int* _order, unsigned int _size)
{
	unsigned int* d_histogramData_ptr;
	unsigned int* d_tmp_ptr;
	unsigned int* d_reference_ptr;
	unsigned int* d_order_ptr;
	unsigned int mem_size = sizeof(unsigned int)*_size;

	hipMalloc(&d_histogramData_ptr, mem_size);
	hipMalloc(&d_tmp_ptr, mem_size);
	hipMalloc(&d_reference_ptr, mem_size);
	hipMalloc(&d_order_ptr,mem_size);

	hipMemcpy(d_histogramData_ptr, _histogramData, mem_size, hipMemcpyHostToDevice);
	hipMemcpy(d_reference_ptr, _reference, mem_size, hipMemcpyHostToDevice);
	hipMemcpy(d_order_ptr, _reference, mem_size, hipMemcpyHostToDevice);

	thrust::device_ptr<unsigned int> d_histogramData(d_histogramData_ptr);
	thrust::device_ptr<unsigned int> d_reference(d_reference_ptr);
	thrust::device_ptr<unsigned int> d_order(d_order_ptr);
	thrust::device_ptr<unsigned int> d_tmp(d_tmp_ptr);

	thrust::sort_by_key(d_histogramData, d_histogramData + _size, d_reference, thrust::greater<unsigned int>());

	hipMemcpy(d_tmp_ptr, d_reference_ptr, mem_size, hipMemcpyDeviceToDevice);
	thrust::sort_by_key(d_tmp, d_tmp + _size, d_order);

	hipMemcpy(_histogramData, d_histogramData_ptr, mem_size, hipMemcpyDeviceToHost);
	hipMemcpy(_reference, d_reference_ptr, mem_size, hipMemcpyDeviceToHost);
	hipMemcpy(_order, d_order_ptr, mem_size, hipMemcpyDeviceToHost);


	hipFree(d_histogramData_ptr);
	hipFree(d_reference_ptr);
	hipFree(d_tmp_ptr);
}

extern "C"
unsigned int binarySearch(unsigned int* _data, unsigned int _value, unsigned int _size)
{
	unsigned int begin = 0, end = _size-1;
	unsigned int middle;
	
	while(begin <= end)
	{
		middle = (begin + end)/2;
		if(_data[middle] == _value)
			return middle;
		else
		{
			if(_value > _data[middle])
				begin = middle +1;
			else
				end = middle -1;
		}
	}

	return 0;

}