#include "hip/hip_runtime.h"
#ifndef _LAYOUT_KERNEL_CU_
#define _LAYOUT_KERNEL_CU_

#include <GL/glew.h>
#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

//#include <cutil_inline.h>
#include "datamanager.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h> 

using namespace VIS;
#define BLOCK_SIZE 512

__device__ float addGraDir(int& id, float* dir, float* baryCenter,float& graFactor,float& repFactor,float& attExponent,t_GraphNode* node)
{
	float dist =0.0;
	float diff =0.0;
	diff  = node[id].pos[0]-baryCenter[0];
	dist += diff*diff;
	diff  = node[id].pos[1]-baryCenter[1];
	dist += diff*diff;
	dist = sqrtf(dist);

	float tmp  = graFactor*repFactor*node[id].weight*pow(dist,attExponent-2);
	dir[0] += (baryCenter[0] - node[id].pos[0])*tmp;
	dir[1] += (baryCenter[1] - node[id].pos[1])*tmp;
	return tmp*fabs(attExponent-1.0);
}

__device__ float  getDistance(int& id1,int& id2,t_GraphNode *node)
{
	float dist =0.0;
	float diff;
	diff = node[id1].pos[0] - node[id2].pos[0];
	dist +=diff*diff;
	diff = node[id1].pos[1] - node[id2].pos[1];
	dist +=diff*diff;
	return sqrtf(dist);
}




__device__ float addAttDir(int &id,int& numNode, float* dir, t_GraphNode* node, t_GraphEdge* edge, float& attExponent)
{
	float dir2 =0.0;
	int index = node[id].begin; 

	for( int i=0;i<node[id].numEdge;i++)
	{
		float dist = getDistance(id, edge[index].node,node);
		if(dist ==0.0)
			continue;

		float tmp = edge[index].weight*pow(dist,attExponent-2);

		dir2 += tmp* fabs(attExponent-1);

		dir[0] += (node[edge[index].node].pos[0] - node[id].pos[0])*tmp ;
		dir[1] += (node[edge[index].node].pos[1] - node[id].pos[1])*tmp ;
		index++;
	}
	return dir2;
}



__device__ float addRepDir(int& id, int& numNode, float& repFactor, float& repExponent,float* dir, t_GraphNode* node)
{
	if(node[id].weight ==0.0)
		return 0.0;

	float dir2=0.0;
	for(int i=0;i< numNode;i++)
	{
		if(i == id || node[i].weight ==0.0)
			continue;

		float dist = getDistance(i,id,node);
		if(dist ==0.0)
			continue;

		float tmp = repFactor*node[i].weight*node[id].weight*pow(dist,repExponent-2);

		dir2 += tmp* fabs(repExponent-1);
		dir[0] -= (node[i].pos[0] - node[id].pos[0])*tmp;
		dir[1] -= (node[i].pos[1] - node[id].pos[1])*tmp;
	}
	return dir2;
}


__device__ void getDirection(int id,float* dir, int& numNode,float& repFactor,float& repExponent, float& graFactor, float& attExponent, float* baryCenter, t_GraphNode* node,t_GraphEdge* edge)
{
	dir[0] = dir[1]=0.0;
	float dir2=0;

	dir2 = addRepDir(id, numNode, repFactor, repExponent, dir,node);
	dir2 += addAttDir(id,numNode, dir,  node,edge,  attExponent);
	dir2 += addGraDir(id, dir, baryCenter, graFactor, repFactor, attExponent,node);

	if(fabs(dir2)<0.00001)
	{
		if(dir2>0)
			dir2 = 0.00001;
		else
			dir2 = -0.00001;
	}

	dir[0] /= dir2;
	dir[1] /= dir2;
}



__global__ void ForceDirect(t_LayoutParamter* parmeter,t_GraphNode* graph_node,t_GraphEdge* graph_edge)
{
	float attExponent = parmeter[0].attExponent;
	float graFactor = parmeter[0].graFactor;
	float repFactor = parmeter[0].repFactor;
	float repExponent = parmeter[0].repExponent;
	int numNode = parmeter[0].numNode;



	int id  = blockIdx.x*blockDim.x + threadIdx.x;
	if(id>= numNode)
		return;

	if(graph_node[id].stable==true)
		return;


	float bestDir[2];
	float oldPos[2];
	bestDir[0] = bestDir[1] = 0.0;
	getDirection(id,bestDir,numNode,repFactor,repExponent,graFactor,attExponent,parmeter[0].baryCenter,graph_node,graph_edge);

	oldPos[0] = graph_node[id].pos[0];
	oldPos[1] = graph_node[id].pos[1];

	//int bestMultiple =0;

	graph_node[id].pos[0] = oldPos[0] + bestDir[0];//*multiple;
	graph_node[id].pos[1] = oldPos[1] + bestDir[1];//*multiple;

	if(fabs(graph_node[id].pos[0])>=parmeter[0].range)
	{ 
		if(graph_node[id].pos[0]>0)
			graph_node[id].pos[0] =parmeter[0].range;
		else
			graph_node[id].pos[0] =0-parmeter[0].range;

	}
	if (fabs(graph_node[id].pos[1])>=parmeter[0].range)
	{
		if(graph_node[id].pos[1]>0)
			graph_node[id].pos[1] =parmeter[0].range;
		else
			graph_node[id].pos[1] =0-parmeter[0].range;
	}


	__syncthreads(); 
}



extern "C"
void kernel_function(t_LayoutParamter* layoutParameter,t_GraphNode* graph_node,t_GraphEdge* graph_edge)
{
	t_LayoutParamter*	d_layoutParameter;
	t_GraphNode*		d_graphNode;
	t_GraphEdge*		d_graphEdge;

	hipMalloc((void**)&d_layoutParameter, sizeof(t_LayoutParamter));
	hipMalloc((void**)&d_graphNode, sizeof( t_GraphNode )*(layoutParameter[0].numNode));
	hipMalloc((void**)&d_graphEdge, sizeof( t_GraphEdge )*(layoutParameter[0].numEdge*2));

	int blocks = layoutParameter[0].numNode/BLOCK_SIZE;
	if(layoutParameter[0].numNode%BLOCK_SIZE!=0)
		blocks++;


	hipMemcpy(d_layoutParameter,layoutParameter,sizeof(t_LayoutParamter),hipMemcpyHostToDevice);
	hipMemcpy(d_graphNode, graph_node,sizeof(t_GraphNode)*layoutParameter[0].numNode,hipMemcpyHostToDevice);
	hipMemcpy(d_graphEdge, graph_edge,sizeof(t_GraphEdge)*layoutParameter[0].numEdge*2,hipMemcpyHostToDevice);
	ForceDirect<<<blocks,BLOCK_SIZE>>>(d_layoutParameter, d_graphNode, d_graphEdge);
	hipMemcpy(graph_node,d_graphNode,sizeof(t_GraphNode)*layoutParameter[0].numNode,hipMemcpyDeviceToHost);


	hipFree(d_layoutParameter);
	hipFree(d_graphNode);
	hipFree(d_graphEdge);


}
#endif